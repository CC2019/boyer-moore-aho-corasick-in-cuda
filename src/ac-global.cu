#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include<fstream>
#include<iostream>
#include <sstream>
#include<queue>
using namespace std;

# define M 3
# define D 6



const int MAXS = M*D + 1;
 
const int MAXC = 26;
 
unsigned int out[MAXS];
 
unsigned int f[MAXS]; 

int g[MAXS][MAXC];
texture<int, hipTextureType2D> tex_state_transition;
texture<unsigned int, hipTextureType1D> tex_state_supply;
texture<unsigned int, hipTextureType1D> tex_state_final;


static void checkCUDAError(const char *msg) {

	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline static void __checkCudaErrors(hipError_t err, const char *file,
		const int line) {

	if (hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line,
				(int) err, hipGetErrorString(err));
		exit(-1);
	}
}


int buildMatchingMachine(string arr[], int k)
{
    // Initialize all values in output function as 0.
    memset(out, 0, sizeof out);
 
    // Initialize all values in goto function as -1.
    memset(g, -1, sizeof g);
 
    // Initially, we just have the 0 state
    int states = 1;
 
    // Construct values for goto function, i.e., fill g[][]
    // This is same as building a Trie for arr[]
    for (int i = 0; i < k; ++i)
    {
        const string &word = arr[i];
        int currentState = 0;
 
        // Insert all characters of current word in arr[]
        for (int j = 0; j < word.size(); ++j)
        {
            int ch = word[j] - 'A';
 
            // Allocate a new node (create a new state) if a
            // node for ch doesn't exist.
            if (g[currentState][ch] == -1)
                g[currentState][ch] = states++;
 
            currentState = g[currentState][ch];
        }
 
        // Add current word in output function
        out[currentState] += 1;
    }
 
    // For all characters which don't have an edge from
    // root (or state 0) in Trie, add a goto edge to state
    // 0 itself
    for (int ch = 0; ch < MAXC; ++ch)
        if (g[0][ch] == -1)
            g[0][ch] = 0;
 
    // Now, let's build the failure function
 
    // Initialize values in fail function
    memset(f, -1, sizeof f);
 
    // Failure function is computed in breadth first order
    // using a queue
    queue<int> q;
 
     // Iterate over every possible input
    for (int ch = 0; ch < MAXC; ++ch)
    {
        // All nodes of depth 1 have failure function value
        // as 0. For example, in above diagram we move to 0
        // from states 1 and 3.
        if (g[0][ch] != 0)
        {
            f[g[0][ch]] = 0;
            q.push(g[0][ch]);
        }
    }
 
    // Now queue has states 1 and 3
    while (q.size())
    {
        // Remove the front state from queue
        int state = q.front();
        q.pop();
 
        // For the removed state, find failure function for
        // all those characters for which goto function is
        // not defined.
        for (int ch = 0; ch <= MAXC; ++ch)
        {
            // If goto function is defined for character 'ch'
            // and 'state'
            if (g[state][ch] != -1)
            {
                // Find failure state of removed state
                int failure = f[state];
 
                // Find the deepest node labeled by proper
                // suffix of string from root to current
                // state.
                while (g[failure][ch] == -1)
                      failure = f[failure];
 
                failure = g[failure][ch];
                f[g[state][ch]] = failure;
 
                // Merge output values
                out[g[state][ch]] += out[failure];
 
                // Insert the next level node (of Trie) in Queue
                q.push(g[state][ch]);
            }
        }
    }
 
    return states;
}



__global__ void ac_kernel1 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int effective_pitch = pitch / sizeof ( int );
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x;
	int stopThread;
	if( blockIdx.x == numBlocks -1 && threadIdx.x==blockDim.x-1)
		stopThread = n - 1;	
	else stopThread = startThread + charactersPerThread + m-1;

	int r = 0, s;
	
	int column;
	
	//cuPrintf("Working from %i to %i chars %i\n", startThread, stopThread, charactersPerThread);
	
	for ( column = startThread; ( column < stopThread && column < n ); column++ ) {

		while ( ( s = d_state_transition[r * effective_pitch + (d_text[column]-(unsigned char)'A')] ) == -1 )
			r = d_state_supply[r];
		r = s;
			
		d_out[idx] += d_state_final[r];
	}
}
__global__ void ac_kernel2 ( unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int numBlocks ) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int charactersPerBlock = n / numBlocks;
	
	int startBlock = blockIdx.x * charactersPerBlock;
	int stopBlock = startBlock + charactersPerBlock;
	
	int charactersPerThread = ( stopBlock - startBlock ) / blockDim.x;
	
	int startThread = startBlock + charactersPerThread * threadIdx.x;
	int stopThread = startThread + charactersPerThread + m - 1;
	
	int r = 0, s;
	
	int column;
	
	for ( column = startThread; ( column < stopThread && column < n ); column++ ) {

		while ( ( s = tex2D ( tex_state_transition, (d_text[column]-(unsigned char)'A'), r ) ) == -1 )
			r = tex1Dfetch ( tex_state_supply, r );
		r = s;
			
		d_out[idx] += tex1Dfetch ( tex_state_final, r );
	}
}



void cuda_ac1 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {

	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 8, numThreadsPerBlock = 1024;

	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;
	
	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//cudaPrintfInit();
	
	//Executing kernel in the device
	ac_kernel1<<<dimGrid, dimBlock>>>( d_state_transition, d_state_supply, d_state_final, d_text, d_out, pitch, m, n, p_size, alphabet, numBlocks );
	checkCUDAError("kernel invocation");
	
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();

	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );
	
	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
	
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  		
	printf ("Global Memory Kernel 1 matches \t%i\t time \t%fms\n", matches, time);
		
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}


void cuda_ac2 ( int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {


	//Pointer for device memory
	int *d_state_transition;
	unsigned int *d_state_supply, *d_state_final, *d_out;
	
	unsigned char *d_text;

	size_t pitch;
	
	int numBlocks = 8, numThreadsPerBlock = 1024;
	dim3 dimGrid ( numBlocks );
	dim3 dimBlock ( numThreadsPerBlock );
	
	if ( n < numBlocks * numThreadsPerBlock * m ) {
		printf("The text size is too small\n");
		exit(1);
	}
	
	//Allocate host memory for results array
	unsigned int *h_out = ( unsigned int * ) malloc ( numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );
	memset ( h_out, 0, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) );

	//Allocate 1D device memory
	checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipMalloc ( ( void** ) &d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ) ) );
	
	//Allocate 2D device memory
	checkCudaErrors ( hipMallocPitch ( &d_state_transition, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
	
	//Copy 1D host memory to device
	checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_supply, state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_state_final, state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
	checkCudaErrors ( hipMemcpy ( d_out, h_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

	//Copy 2D host memory to device
	checkCudaErrors ( hipMemcpy2D ( d_state_transition, pitch, state_transition, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
	
	//Bind the preprocessing tables to the texture cache
	hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
	
	checkCudaErrors ( hipBindTexture2D ( 0, tex_state_transition, d_state_transition, desc, alphabet, m * p_size + 1, pitch ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_supply, d_state_supply, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	checkCudaErrors ( hipBindTexture ( 0, tex_state_final, d_state_final, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
	
	//Create timer
	hipEvent_t start, stop;

	float time;

	//Create the timer events
	hipEventCreate ( &start );
	hipEventCreate ( &stop );
	
	//Start the event clock	
	hipEventRecord ( start, 0 );
	
	//Executing kernel in the device
	ac_kernel2<<<dimGrid, dimBlock>>>( d_text, d_out, m, n, p_size, alphabet, numBlocks );
	checkCUDAError("kernel invocation");
	
	hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	
	hipEventElapsedTime ( &time, start, stop );
	
	hipEventDestroy ( start );
	hipEventDestroy ( stop );

	//Get back the results from the device
	hipMemcpy ( h_out, d_out, numBlocks * numThreadsPerBlock * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
   
  	//Look at the results
  	int i, matches = 0;
  	for ( i = 0; i < numBlocks * numThreadsPerBlock; i++ )
  		matches += h_out[i];
  	
	printf ("Texture Memory Kernel matches \t%i\t time \t%fms\n", matches, time);
	
	hipUnbindTexture ( tex_state_transition );
	hipUnbindTexture ( tex_state_supply );
	hipUnbindTexture ( tex_state_final );
	
	//Free host and device memory
	free ( h_out );
	
	hipFree ( d_text );
	hipFree ( d_state_transition );
	hipFree ( d_state_supply );
	hipFree ( d_state_final );
	hipFree ( d_out );
}


int main(){

	string patterns[]={"ATC","GTG","GTC","ATG","CAA","ATT"};
	int k = sizeof(patterns)/sizeof(patterns[0]);
	string text;

	std::ifstream t("data.txt");
	std::stringstream buffer;
	buffer << t.rdbuf();	
	text = buffer.str();
	unsigned char *charText = (unsigned char*)text.c_str();
	
	buildMatchingMachine(patterns, k);

	int *goToTable = (int*)malloc(sizeof(int)*MAXC*MAXS);
	for(int i=0;i<MAXS;i++)
		for(int j=0;j<MAXC;j++)
			goToTable[i*MAXC+j] = g[i][j];

//(int m, unsigned char *text, int n, int p_size, int alphabet, int *state_transition, unsigned int *state_supply, unsigned int *state_final ) {
	
	cuda_ac1(M,charText,text.size(),D,26,goToTable,f,out);
	//cuda_ac2(M,charText,text.size(),D,26,goToTable,f,out);
	
	return 0;
}




