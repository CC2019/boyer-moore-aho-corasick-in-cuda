#include "hip/hip_runtime.h"
#include "project-header.h"

texture<int, hipTextureType2D> tex_go_to_state;
texture<unsigned int, hipTextureType1D> tex_failure_state;
texture<unsigned int, hipTextureType1D> tex_output_state;

__global__ 
void texture_kernel(unsigned char *d_text, unsigned int *d_out, int m, int n, int p_size, int alphabet, int num_blocks ) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int characters_per_block = n / num_blocks;
    
    int start_block = blockIdx.x * characters_per_block;
    int stop_block = start_block + characters_per_block;
    
    int characters_per_thread = ( stop_block - start_block ) / blockDim.x;
    
    int start_thread = start_block + characters_per_thread * threadIdx.x;
    int stop_thread = start_thread + characters_per_thread + m - 1;
    
    int r = 0, s;
    
    int column;
    
    for ( column = start_thread; ( column < stop_thread && column < n ); column++ ) {

        while ( ( s = tex2D ( tex_go_to_state, (d_text[column]-(unsigned char)'A'), r ) ) == -1 )
            r = tex1Dfetch ( tex_failure_state, r );
        r = s;
            
        d_out[idx] += tex1Dfetch ( tex_output_state, r );
    }
}

void texture_memory_wrapper_func(int m, unsigned char *text, int n, int p_size, int alphabet, int *go_to_state, unsigned int *failure_state, unsigned int *output_state ) {

    //Pointer for device memory
    int *d_go_to_state;
    unsigned int *d_failure_state, *d_output_state, *d_out;
    
    unsigned char *d_text;

    size_t pitch;
    
    int num_blocks = 8, num_threads_per_block = 1024;
    dim3 dimGrid ( num_blocks );
    dim3 dimBlock ( num_threads_per_block );
    
    if ( n < num_blocks * num_threads_per_block * m ) {
        printf("The text size is too small\n");
        exit(1);
    }
    
    //Allocate host memory for results array
    unsigned int *h_out = ( unsigned int * ) malloc ( num_blocks * num_threads_per_block * sizeof ( unsigned int ) );
    memset ( h_out, 0, num_blocks * num_threads_per_block * sizeof ( unsigned int ) );

    //Allocate 1D device memory
    checkCudaErrors ( hipMalloc ( ( void** ) &d_text, n * sizeof ( unsigned char ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipMalloc ( ( void** ) &d_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ) ) );
    
    //Allocate 2D device memory
    checkCudaErrors ( hipMallocPitch ( &d_go_to_state, &pitch, alphabet * sizeof ( int ), ( m * p_size + 1 ) ) );
    
    //Copy 1D host memory to device
    checkCudaErrors ( hipMemcpy ( d_text, text, n * sizeof ( unsigned char ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_failure_state, failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_output_state, output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );
    checkCudaErrors ( hipMemcpy ( d_out, h_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ), hipMemcpyHostToDevice ) );

    //Copy 2D host memory to device
    checkCudaErrors ( hipMemcpy2D ( d_go_to_state, pitch, go_to_state, alphabet * sizeof ( int ), alphabet * sizeof ( int ), ( m * p_size + 1 ), hipMemcpyHostToDevice ) );
    
    //Bind the preprocessing tables to the texture cache
    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    
    checkCudaErrors ( hipBindTexture2D ( 0, tex_go_to_state, d_go_to_state, desc, alphabet, m * p_size + 1, pitch ) );
    checkCudaErrors ( hipBindTexture ( 0, tex_failure_state, d_failure_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    checkCudaErrors ( hipBindTexture ( 0, tex_output_state, d_output_state, ( m * p_size + 1 ) * sizeof ( unsigned int ) ) );
    
    //Create timer
    hipEvent_t start, stop;

    float time;

    //Create the timer events
    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    
    //Start the event clock 
    hipEventRecord ( start, 0 );
    
    //Executing kernel in the device
    texture_kernel<<<dimGrid, dimBlock>>>( d_text, d_out, m, n, p_size, alphabet, num_blocks );
    checkCUDAError("kernel invocation");
    
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    
    hipEventElapsedTime ( &time, start, stop );
    
    hipEventDestroy ( start );
    hipEventDestroy ( stop );

    //Get back the results from the device
    hipMemcpy ( h_out, d_out, num_blocks * num_threads_per_block * sizeof ( unsigned int ), hipMemcpyDeviceToHost );
   
    //Look at the results
    int i, matches = 0;
    for ( i = 0; i < num_blocks * num_threads_per_block; i++ )
        matches += h_out[i];
    
    printf ("Texture Memory Kernel matches \t%i\t time \t%fms\n", matches, time);
    
    hipUnbindTexture ( tex_go_to_state );
    hipUnbindTexture ( tex_failure_state );
    hipUnbindTexture ( tex_output_state );
    
    //Free host and device memory
    free ( h_out );
    
    hipFree ( d_text );
    hipFree ( d_go_to_state );
    hipFree ( d_failure_state );
    hipFree ( d_output_state );
    hipFree ( d_out );
}


int main(){

    string patterns[]={"ATC","GTG","GTC","ATG","CAA","ATT"};
    int k = sizeof( patterns )/sizeof( patterns[0] );
    string text;

    ifstream t( "data.txt" );
    stringstream buffer;
    buffer << t.rdbuf();    
    text = buffer.str();
    unsigned char *charText = ( unsigned char* )text.c_str();
    
    buildMatchingMachine(patterns, k);

    int *go_to_table = (int*)malloc( sizeof(int)*MAXC*MAXS );
    
    for(int i=0;i<MAXS;i++)
        for(int j=0;j<MAXC;j++)
            go_to_table[i*MAXC+j] = g[i][j];
    
    texture_memory_wrapper_func( M, charText, text.size(), D, 26, go_to_table, f, out );
    
    return 0;
}